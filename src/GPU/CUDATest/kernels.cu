#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "kernels.h"

__global__ void helloworld(const char *in, char *out, int count)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < count)
    {
		out[i] = in[i] + 1;
    }
}


extern "C" void CallHelloWorld( const char *in, char *out, int count)
{
    helloworld<<<1, count>>>(in, out, count);
}